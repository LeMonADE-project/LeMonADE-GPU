#include "hip/hip_runtime.h"

#include <LeMonADEGPU/utility/GPUConnectionTracker.h>
#include <LeMonADEGPU/utility/GPUConnectionTrackerTendomer.h>
#include <LeMonADEGPU/utility/cudacommon.hpp>
#include <LeMonADEGPU/utility/DeleteMirroredObject.h>
#include <LeMonADEGPU/core/constants.cuh>

using ID_t               = Tracker<uint8_t>::ID_t;
using T_Coordinates      = Tracker< uint8_t >::T_Coordinates;
using T_Coordinate       = Tracker< uint8_t >::T_Coordinate;
using T_BoxSize          = uint64_t;

template< typename T_UCoordinateCuda >
__global__ void kernelTrackBreaks
(
  ID_t           * const dID1      ,
  ID_t           * const dID2      ,
  size_t           const dSize     ,
  int32_t          const dOffsetA  ,
  int32_t          const dOffsetB  ,
  ID_t           * const diNewToi  ,
  T_Coordinates * const __restrict__ dOutputID1,
  T_Coordinates * const dOutputID2,
  typename CudaVec4< T_UCoordinateCuda >::value_type const * const __restrict__ dpPolymerSystem,
  T_Coordinates const * const dpiPolymerSystemSortedVirtualBox 
)
{
  for ( auto i = blockIdx.x * blockDim.x + threadIdx.x;
          i < dSize; i += gridDim.x * blockDim.x )
  {
    auto iMonomer(dID1[i]);
    auto iPartner(dID2[i]);
    if (iPartner == 0 || iMonomer == 0 ) 
    {
      continue; //no Partner found -> go to next Crosslink in the grid 
    }
    iMonomer--;
    iPartner--;
    dID1[i]=0;
    dID2[i]=0;
    auto gID1(iMonomer + dOffsetA);
    auto rsmall = dpPolymerSystem[ gID1 ];
    //cast from T_UCoordinateCuda = uint32_t "down" to int32_t
    T_Coordinates rSorted = { T_Coordinate( rsmall.x ), T_Coordinate( rsmall.y ),
                              T_Coordinate( rsmall.z ), T_Coordinate( rsmall.w )};
    auto nPos = dpiPolymerSystemSortedVirtualBox[ gID1 ];
    rSorted.x += nPos.x * dcBoxX;
    rSorted.y += nPos.y * dcBoxY;
    rSorted.z += nPos.z * dcBoxZ;
    rSorted.w  = diNewToi[gID1];
    dOutputID1[i] = rSorted;

    auto gID2(iPartner+dOffsetB);
    rsmall = dpPolymerSystem[ gID2 ];
    //cast from T_UCoordinateCuda = uint32_t "down" to int32_t
    rSorted = { T_Coordinate( rsmall.x ), T_Coordinate( rsmall.y ),
                T_Coordinate( rsmall.z ), T_Coordinate( rsmall.w )};
    nPos = dpiPolymerSystemSortedVirtualBox[ gID2 ];
    rSorted.x += nPos.x * dcBoxX;
    rSorted.y += nPos.y * dcBoxY;
    rSorted.z += nPos.z * dcBoxZ;
    rSorted.w  = ( (diNewToi[gID2]+1)<<1 )+0;
    dOutputID2[i] = rSorted;

  }
}
template< typename T_UCoordinateCuda > 
void TrackerTendomer<T_UCoordinateCuda>::trackBreaks( ID_t * const ID1     ,
			   ID_t * const ID2     ,
			   size_t const size    ,     
			   ID_t * const diNewToi,
			   int32_t const offsetA,
			   int32_t const offsetB,
			   uint32_t const mAge  ,
         MirroredVector< T_UCoordinatesCuda >const * const mPolymerSystemSorted , 
         MirroredVector< T_Coordinates      >const * const mviPolymerSystemSortedVirtualBox  )
{
  auto nThreads(256);
  auto nBlocks(ceilDiv(size,nThreads));
  kernelTrackBreaks<T_UCoordinateCuda><<<nBlocks,nThreads,0, mStream>>>(
  ID1, 
  ID2, 
  size, 
  offsetA,
  offsetB,
  diNewToi, 
  BondHistoryID1->gpu + counter*nIDs,
  BondHistoryID2->gpu + counter*nIDs,
  mPolymerSystemSorted->gpu,
  mviPolymerSystemSortedVirtualBox->gpu
  );  
  age.push_back(mAge);
  increaseCounter();	
  if(counter == bufferSize )
  dumpReactions();
}



/**
 * @brief calculates the minimal distances of images for one component 
 * @return int 
 * @param x1 absolute coordinate
 * @param x2 absolute coordinate
 * @param LatticeSize size of the box in the direction of the given coordinates
 */
 template <class T >
__device__ __host__  T inline MinImageDistanceComponentForPowerOfTwo(const T x, const uint32_t latticeSize ){
  //this is only valid for absolute coordinates
  uint32_t latticeSizeM1(latticeSize-1);
  return ( ((x&latticeSizeM1) < (latticeSize/2)) ? (x & latticeSizeM1) :  -(-x & latticeSizeM1));
}

template <class T >
__device__  T_Coordinates inline calcVector(const T vec, const T_Coordinates pVec ){
  //cast from T_UCoordinateCuda = uint32_t "down" to int32_t
  T_Coordinates rSorted = { T_Coordinate( vec.x ), T_Coordinate( vec.y ),T_Coordinate( vec.z ), T_Coordinate( vec.w )};
  rSorted.x += pVec.x * dcBoxX;
  rSorted.y += pVec.y * dcBoxY;
  rSorted.z += pVec.z * dcBoxZ;
  return rSorted; 
}

__device__ T_Coordinates inline MinImageVector(const T_Coordinates vec1, const T_Coordinates vec2) {

  T_Coordinates vec={   MinImageDistanceComponentForPowerOfTwo( vec2.x-vec1.x, dcBoxX ), 
                        MinImageDistanceComponentForPowerOfTwo( vec2.y-vec1.y, dcBoxY ), 
                        MinImageDistanceComponentForPowerOfTwo( vec2.z-vec1.z, dcBoxZ ), 
                        T_Coordinate(0)
                    };
  return vec;

}
__device__ T_Coordinates addVectors(const T_Coordinates vec1, const T_Coordinates vec2)
{
 return T_Coordinates{vec1.x+vec2.x, vec1.y+vec2.y,vec1.z+vec2.z,vec1.w+vec2.w,};
}
__device__ T_Coordinates substractVectors(const T_Coordinates vec1, const T_Coordinates vec2)
{
 return T_Coordinates{vec1.x-vec2.x, vec1.y-vec2.y,vec1.z-vec2.z,vec1.w-vec2.w,};
}

template< typename T_UCoordinateCuda >
__global__ void kernelTrackConnections
(
  ID_t           * const dID1      ,
  ID_t           * const dID2      ,
  size_t           const dSize     ,
  int32_t          const dOffsetA  ,
  int32_t          const dOffsetB  ,
  ID_t           * const diNewToi  ,
  ID_t           * const diToiNew  ,
  T_Coordinates  * const dOutputID1,
  T_Coordinates  * const dOutputID2,
  ID_t           * const dChainID  ,
  typename CudaVec4< T_UCoordinateCuda >::value_type const * const __restrict__ dpPolymerSystem,
  T_Coordinates const * const dpiPolymerSystemSortedVirtualBox,
  ID_t           * const dMidToNid,
  ID_t           * const dNidToMid,
  ID_t           * const dNidToNid,
  ID_t           * const dNidToCid
)
{
  for ( auto i = blockIdx.x * blockDim.x + threadIdx.x;
          i < dSize; i += gridDim.x * blockDim.x )
    { 
      auto iMonomer(dID1[i]);
      auto iPartner(dID2[i]);
      dID1[i]=0;
      dID2[i]=0;
      if (iPartner == 0 || iMonomer == 0 ) {
	      continue; //no Partner found -> go to next Crosslink in the grid 
      }

      iMonomer--;
      iPartner--;
      
      //Crosslink1:
      auto gID1(iMonomer + dOffsetA);
      T_Coordinates rCrosslink1( calcVector(dpPolymerSystem[ gID1 ],dpiPolymerSystemSortedVirtualBox[ gID1 ]  ) ); 
      rCrosslink1.w  = diNewToi[gID1];

      dOutputID1[i] = rCrosslink1;
      ////////////////////////////
      // Chain monomer 1 : 
      auto gID2(iPartner+dOffsetB); // one chain end monomer, global id 
      auto gID2Old(diNewToi[gID2]);
      auto reducedMonChainID1(dMidToNid[gID2Old]); // reduced chain id -> chain start 
      auto reducedMonChainID2(dNidToNid[reducedMonChainID1 ]); // reduced chain id -> chain end 
      auto crosslinkID(dNidToCid[reducedMonChainID2]); // second cross link id, id zero there is no cross link connected to the first cross link, global id + 1
      auto gMonoOnChain2(diToiNew[dNidToMid[reducedMonChainID2]] );// second  chain end monomer, global id 
      dChainID[i]=(reducedMonChainID1-(reducedMonChainID1%2) )/2 ; // chain ID where the first monomer is attached to 
      // printf("ChainID=%d  rCID=%d %d %d ID1=%d ID2=%d \n", dChainID[i], reducedMonChainID1,reducedMonChainID2,crosslinkID ,  diNewToi[gID1], diNewToi[gID2] );
      T_Coordinates rRefoldCrosslink2={ 0 , 0 ,  0 ,  3 }; // 3=(( 0+1)<<1 )+1 ; 
      dNidToCid[reducedMonChainID1]=gID1+1;
      if( crosslinkID >0 ){
        crosslinkID= crosslinkID-1;
        // The cross links and the chains are connected across periodic images. Thus the bonds can be "bond+multiple of box size". 
        // To reduce this to the real value, we calculate the bond1 from the cross link to the chain start (reduce to MIC) 
        // , add the vector from the end-to-end vector of the chain and the vector of the chain end to the second cross link(MIC again).
        // printf("gID2=%d gMonoOnChain2=%d crosslinkID=%d\n",gID2, gMonoOnChain2, crosslinkID);
        // position of the chain start 
        T_Coordinates rChain1(calcVector(dpPolymerSystem[ gID2 ], dpiPolymerSystemSortedVirtualBox[ gID2 ]));
        // position of the chain end 
        T_Coordinates rChain2(calcVector(dpPolymerSystem[ gMonoOnChain2 ], dpiPolymerSystemSortedVirtualBox[ gMonoOnChain2 ]));
        // position of the connected cross link 
        T_Coordinates rCrosslink2(calcVector(dpPolymerSystem[ crosslinkID ], dpiPolymerSystemSortedVirtualBox[ crosslinkID ]));
        //calculate the refolded position of the second cross link 
        rRefoldCrosslink2=( rCrosslink1 + MinImageVector(rCrosslink1,rChain1) + substractVectors(rChain2,rChain1) + MinImageVector( rChain2,rCrosslink2)  );
        // rRefoldCrosslink2=rCrosslink2;
        rRefoldCrosslink2.w  = ( (diNewToi[crosslinkID]+1)<<1 )+1;
      }
      dOutputID2[i] = rRefoldCrosslink2;
      // printf ("i=%d out1=%d (%d,%d,%d)  cID=%d out2=%d\n", i, dOutputID1[i].w,dOutputID1[i].x, dOutputID1[i].y,dOutputID1[i].z, dChainID[i],dOutputID2[i].w);
    }
}
__global__ void kernelControllSettings(
ID_t           * const dMidToNid,
ID_t           * const dNidToMid,
ID_t           * const dNidToNid,
ID_t           * const dNidToCid
)
{
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (dMidToNid[i] >0 || ( i==0 && dMidToNid[i]==0) )
    printf("i=%d M->N=%d N->M=%d N->N=%d N->C=%d N->C=%d \n",i, dMidToNid[i], dNidToMid[dMidToNid[i]], dNidToNid[dMidToNid[i]],dNidToCid[dMidToNid[i]], dNidToCid[dNidToNid[dMidToNid[i]]] );
}
__global__ void kernelPrintBondHist(
  T_Coordinates  * const dOutputID1,
  T_Coordinates  * const dOutputID2,
  size_t           const dSize     
)
{  for ( auto i = blockIdx.x * blockDim.x + threadIdx.x;
    i < dSize; i += gridDim.x * blockDim.x ){
      if (dOutputID1[i].w!=0)
      printf("Hist i=%d ID1=%d ID2=%d\n", i , dOutputID1[i].w, dOutputID2[i].w );
   }

}
template< typename T_UCoordinateCuda > 
void TrackerTendomer<T_UCoordinateCuda>::trackConnections( 
        ID_t * const ID1     ,
        ID_t * const ID2     ,
        size_t const size    ,     
        ID_t * const diNewToi,
        ID_t * const diToiNew,
        int32_t const offsetA,
        int32_t const offsetB,
        uint32_t const mAge,
        MirroredVector< T_UCoordinatesCuda >const * const  mPolymerSystemSorted , 
        MirroredVector< T_Coordinates      >const * const mviPolymerSystemSortedVirtualBox
      )
{
  auto nThreads(256);	
  auto nBlocks(ceilDiv(size,nThreads));
//   std::cout << "Tracker::trackConnections:   mAge= " << mAge  << " size= " << size <<std::endl;
//   kernelPrintBondHist<<<nBlocks,nThreads,0,mStream>>>(    BondHistoryID1->gpu + counter*nIDs,
//     BondHistoryID2->gpu + counter*nIDs,size) ;
  CUDA_ERROR( hipStreamSynchronize( mStream ) );
  kernelTrackConnections<T_UCoordinateCuda><<<nBlocks,nThreads,0, mStream>>>(
  ID1, 
  ID2, 
  size, 
  offsetA,
  offsetB,
  diNewToi, 
  diToiNew, 
  BondHistoryID1->gpu + counter*nIDs,
  BondHistoryID2->gpu + counter*nIDs,
  mChainID->gpu       + counter*nIDs,
  mPolymerSystemSorted->gpu,
  mviPolymerSystemSortedVirtualBox->gpu,
  mMidToNid->gpu,
  mNidToMid->gpu,
  mNidToNid->gpu,
  mNidToCid->gpu
  );
  CUDA_ERROR( hipStreamSynchronize( mStream ) );
  age.push_back(mAge);
  increaseCounter();
  if(counter == bufferSize ) 
    dumpReactions();
}
template< typename T_UCoordinateCuda > 
TrackerTendomer<T_UCoordinateCuda>::TrackerTendomer():
bufferSize      ( 0             ), 
nIDs            ( 0             ), 
mStream         ( hipStream_t()), 
counter         ( 0             ), 
IDoffset        ( 0             ), 
BaseClass       (               ),
mMidToNid       ( NULL          ),
mNidToMid       ( NULL          ),
mNidToNid       ( NULL          ),
mNidToCid       ( NULL          ),
BondHistoryID1  ( NULL          ),
BondHistoryID2  ( NULL          ),
mChainID        ( NULL          )
{}

template< typename T_UCoordinateCuda > 
void TrackerTendomer<T_UCoordinateCuda>::init(uint32_t bufferSize_, uint32_t nIDs_, hipStream_t mStream_,
    T_BoxSize const boxX_,
    T_BoxSize const boxY_,
    T_BoxSize const boxZ_,
    uint32_t nMonomerPerChain_,
    uint32_t nTendomers_)
{
      nMonomerPerChain=nMonomerPerChain_;
      nTendomers=nTendomers_;
      bufferSize=bufferSize_; 
      nIDs=nIDs_; 
      mStream=mStream_;
      std::cout << "Tracker::init: \nnTendomers=" << nTendomers<<"\n"
                << "nMonomerPerChain=" <<nMonomerPerChain<<"\n"
                << "bufferSize=" <<bufferSize<<"\n"
                << "nIDs=" <<nIDs<<"\n"
                << "mStream=" <<mStream<<"\n";
      BaseClass::setInformationSize(11);
      BaseClass::addComment("MCS Bond/Break ChainID ID1 Position1 ID2 Position2 ");
      std::cout << "Tracker::init: each BondHistory can take " 
                << bufferSize*nIDs << " number of elements with " 
                << bufferSize*nIDs *sizeof(T_Coordinates)/1024.<< " kB \n";
      BondHistoryID1 = new MirroredVector< T_Coordinates >( bufferSize*nIDs, mStream ); //essentially the ids of the first monomer and its positions
      BondHistoryID2 = new MirroredVector< T_Coordinates >( bufferSize*nIDs, mStream ); //essentially the ids of the second monomer and its positions
      mChainID       = new MirroredVector<          ID_t >( bufferSize*nIDs, mStream ); //the chain id between monomer one and two 
      mMidToNid      = new MirroredVector<          ID_t >( nMonomerPerChain*2*nTendomers, mStream );
      mNidToMid      = new MirroredVector<          ID_t >( 2*nTendomers, mStream );
      mNidToNid      = new MirroredVector<          ID_t >( 2*nTendomers, mStream );
      mNidToCid      = new MirroredVector<          ID_t >( 2*nTendomers, mStream );
      for ( size_t i=0; i < nTendomers; i++){
        //MonID to ChainID
        mMidToNid->host[i*2*nMonomerPerChain]=i*2;
        mMidToNid->host[(i*2+1)*nMonomerPerChain]=i*2+1;
        //ChainID to MonID
        mNidToMid->host[i*2]=i*2*nMonomerPerChain;
        mNidToMid->host[i*2+1]=(i*2+1)*nMonomerPerChain;
        //ChainID to ChainID : start to end
        mNidToNid->host[i*2]=i*2+1;
        mNidToNid->host[i*2+1]=i*2;
    
        mNidToCid->host[i*2]=0;
        mNidToCid->host[i*2+1]=0;
    
        if( i <20 )
        {
            std::cout << "i=" << i <<  " "
                      << "M->N(" << i*nMonomerPerChain << ")=" << mMidToNid->host[i*nMonomerPerChain] << " "
                      << "N->M(" << i                  << ")="<< mNidToMid->host[i] << " "
                      << "N->N(" << i                  << ")="<< mNidToNid->host[i] << " "
                      << "N->C(" << i                  << ")="<< mNidToCid->host[i] << " "
                      << std::endl;
        }
      }
      { decltype( dcBoxX      ) x = boxX_     ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxX     ), &x, sizeof(x) ) ); }
      { decltype( dcBoxY      ) x = boxY_     ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxY     ), &x, sizeof(x) ) ); }
      { decltype( dcBoxZ      ) x = boxZ_     ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxZ     ), &x, sizeof(x) ) ); }
}

template< typename T_UCoordinateCuda > 
void TrackerTendomer<T_UCoordinateCuda>::addCrosslinkConnection(uint32_t chainEndID_, uint32_t crosslinkID_)
{
  mNidToCid->host[mMidToNid->host[chainEndID_]]=crosslinkID_; 
}
template< typename T_UCoordinateCuda > 
void TrackerTendomer<T_UCoordinateCuda>::pushToGPU(ID_t const * const miToiNew){
  mMidToNid->push();
  mNidToMid->push();
  mNidToNid->push();
  for( size_t i=0; i <mNidToCid->nElements; i++){
    if (mNidToCid->host[i] > 0 )
      mNidToCid->host[i]= miToiNew[mNidToCid->host[i] ] +1 ;
  }
  kernelControllSettings<<<1,256>>>(  mMidToNid->gpu,
    mNidToMid->gpu,
    mNidToNid->gpu,
    mNidToCid->gpu);
  mNidToCid->push();
  for (uint32_t j=0 ; j < bufferSize ; j ++ ) {
    for(uint32_t i =0 ; i < nIDs; i ++){
      auto index(i+nIDs*j);
      BondHistoryID1->host[index].w=0;
      BondHistoryID1->host[index].x=0;
      BondHistoryID1->host[index].y=0;
      BondHistoryID1->host[index].z=0;
      BondHistoryID2->host[index].w=0;
      BondHistoryID2->host[index].x=0;
      BondHistoryID2->host[index].y=0;
      BondHistoryID2->host[index].z=0;
      mChainID->host[index]=0;
    }
  }
  BondHistoryID1->push();
  BondHistoryID2->push();
  mChainID -> push();
  CUDA_ERROR( hipStreamSynchronize( mStream ) );
}
template< typename T_UCoordinateCuda > 
void TrackerTendomer<T_UCoordinateCuda>::increaseCounter()
{
  counter++;
}

template< typename T_UCoordinateCuda > 
void TrackerTendomer<T_UCoordinateCuda>::dumpReactions()
{
  CUDA_ERROR( hipStreamSynchronize( mStream ) );
  BondHistoryID1->popAsync();
  BondHistoryID2->popAsync();
  mChainID->popAsync();
  CUDA_ERROR( hipStreamSynchronize( mStream ) );
  BaseClass::setBufferSize(bufferSize);
  for (uint32_t j=0 ; j < counter ; j ++ ) {
    int32_t currentAge(age[j]);
    for(uint32_t i =0 ; i < nIDs; i ++)
    {

      auto index(i+nIDs*j);
      auto Mon1(BondHistoryID1->host[index]);
      auto Mon2(BondHistoryID2->host[index]);
      auto MonID1(Mon1.w); 
      auto MonID2(Mon2.w); 
      
      if( MonID2  > 0 )
      {
        std::vector<int32_t> vec;
        vec.push_back(currentAge); //time 
        vec.push_back( MonID2 & 1 ); // either 0 or 1 for remove or add 
        vec.push_back(mChainID->host[index]);
        MonID2 = (MonID2 >> 1) -1;
        if (MonID1 > MonID2 ) {
          vec.push_back(MonID1); 
          vec.push_back(Mon1.x);
          vec.push_back(Mon1.y);
          vec.push_back(Mon1.z);
          vec.push_back(MonID2); 
          vec.push_back(Mon2.x);
          vec.push_back(Mon2.y);
          vec.push_back(Mon2.z);
        }else {
          vec.push_back(MonID2); 
          vec.push_back(Mon2.x);
          vec.push_back(Mon2.y);
          vec.push_back(Mon2.z);
          vec.push_back(MonID1); 
          vec.push_back(Mon1.x);
          vec.push_back(Mon1.y);
          vec.push_back(Mon1.z);
        }
        // vec.push_back(std::min(Mon1,Mon2));   
        // vec.push_back(std::max(Mon1,Mon2));
        BaseClass::addConnection(vec);
        BondHistoryID1->host[index].w=0;
        BondHistoryID1->host[index].x=0;
        BondHistoryID1->host[index].y=0;
        BondHistoryID1->host[index].z=0;
        BondHistoryID2->host[index].w=0;
        BondHistoryID2->host[index].x=0;
        BondHistoryID2->host[index].y=0;
        BondHistoryID2->host[index].z=0;
        mChainID->host[index]=0;
      }
    }
  }
  BaseClass::dumpReactions();
  counter=0;
  age.resize(0);
  BondHistoryID1->push();
  BondHistoryID2->push();
  mChainID->push();
  
}
template class TrackerTendomer< uint8_t  >;
template class TrackerTendomer< uint16_t >;
template class TrackerTendomer< uint32_t >;
template class TrackerTendomer<  int16_t >;
template class TrackerTendomer<  int32_t >;
