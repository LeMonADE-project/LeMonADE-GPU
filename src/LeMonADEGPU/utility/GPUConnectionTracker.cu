#include "hip/hip_runtime.h"

#include <LeMonADEGPU/utility/GPUConnectionTracker.h>

using ID_t = Tracker::ID_t;

__global__ void kernelTrackBreaks
(
  ID_t           * const dID1      ,
  ID_t           * const dID2      ,
  ID_t             const dSize     ,
  int32_t          const dOffsetA  ,
  int32_t          const dOffsetB  ,
  ID_t           * const diNewToi  ,
  ID_t           * const dOutputID1,
  ID_t           * const dOutputID2 
)
{
  for ( auto i = blockIdx.x * blockDim.x + threadIdx.x;
          i < dSize; i += gridDim.x * blockDim.x )
  {
    auto iMonomer(dID1[i]);
    auto iPartner(dID2[i]);
    if (iPartner == 0 || iMonomer == 0 ) 
    {
      continue; //no Partner found -> go to next Crosslink in the grid 
    }
    iMonomer--;
    iPartner--;
    dID1[i]=0;
    dID2[i]=0;
    dOutputID1[i] = diNewToi[iMonomer + dOffsetA ];
    dOutputID2[i] = ( (diNewToi[iPartner+dOffsetB]+1)<<1 )+0;
//     output[miNewToi[iMonomer-1]-dOffsetA]= ( (miNewToi[iPartner-1]-dOffsetB+1)<<1 )+0;
//     printf("Breaks index=%d %d Id1=%d Id2=%d\n ",diNewToi[iMonomer+dOffsetA], diNewToi[iPartner+dOffsetB], iMonomer+dOffsetA,iPartner+dOffsetB );
  }
}

void Tracker::trackBreaks( ID_t * const ID1     ,
			   ID_t * const ID2     ,
			   ID_t   const size    ,     
			   ID_t * const diNewToi,
			   int32_t const offsetA,
			   int32_t const offsetB,
			   uint32_t const mAge  )
{
  auto nThreads(256);
  auto nBlocks(ceilDiv(size,nThreads));
  kernelTrackBreaks<<<nBlocks,nThreads,0, mStream>>>(
  ID1, 
  ID2, 
  size, 
  offsetA,
  offsetB,
  diNewToi, 
  BondHistoryID1->gpu + counter*nIDs,
  BondHistoryID2->gpu + counter*nIDs
  );  
  age.push_back(mAge);
  increaseCounter();	
  if(counter == bufferSize )
  dumpReactions();
}
__global__ void kernelTrackConnections
(
  ID_t           * const dID1      ,
  ID_t           * const dID2      ,
  ID_t             const dSize     ,
  int32_t          const dOffsetA  ,
  int32_t          const dOffsetB  ,
  ID_t           * const diNewToi  ,
  ID_t           * const dOutputID1,
  ID_t           * const dOutputID2
)
{
  for ( auto i = blockIdx.x * blockDim.x + threadIdx.x;
          i < dSize; i += gridDim.x * blockDim.x )
    {
      auto iMonomer(dID1[i]);
      auto iPartner(dID2[i]);
      dID1[i]=0;
      dID2[i]=0;
      if (iPartner == 0 || iMonomer == 0 ) 
      {
	continue; //no Partner found -> go to next Crosslink in the grid 
      }
      iMonomer--;
      iPartner--;
      dOutputID1[i] = diNewToi[iMonomer + dOffsetA ];
      dOutputID2[i] = ( (diNewToi[iPartner+dOffsetB]+1)<<1 )+1;
//       output[miNewToi[iMonomer-1+offsetA]]= ( (miNewToi[iPartner-1+offsetB]+1)<<1 )+1;
//       printf("Bonds Mon1 = %d  Mon2 = %d Id1=%d Id2=%d %d %d  \n ", iMonomer+dOffsetA, iPartner+dOffsetB, diNewToi[iMonomer+dOffsetA],diNewToi[iPartner+dOffsetB], dOffsetA,dOffsetB );
    }
}
void Tracker::trackConnections( ID_t * const ID1     ,
                                ID_t * const ID2     ,
                                ID_t   const size    ,     
				ID_t * const diNewToi,
				int32_t const offsetA,
				int32_t const offsetB,
				uint32_t const mAge)
{
  auto nThreads(256);	
  auto nBlocks(ceilDiv(size,nThreads));
//   std::cout << "Tracker::trackConnections: offsetA= "<< offset << " mAge= " << mAge  << " size= " << size <<std::endl;
  kernelTrackConnections<<<nBlocks,nThreads,0, mStream>>>(
  ID1, 
  ID2, 
  size, 
  offsetA,
  offsetB,
  diNewToi, 
  BondHistoryID1->gpu + counter*nIDs,
  BondHistoryID2->gpu + counter*nIDs
  );
  age.push_back(mAge);
  increaseCounter();
  if(counter == bufferSize )
  dumpReactions();
}

void Tracker::init(uint32_t bufferSize_, uint32_t nIDs_, hipStream_t mStream_)
{
  bufferSize=bufferSize_; nIDs=nIDs_; mStream=mStream_;
  BaseClass::setInformationSize(4);
  BaseClass::addComment("MCS Bond/Break ID1 ID2");
  std::cout << "Tracker::init: each BondHistory can take " 
            << 2*bufferSize*nIDs << " number of elements with " 
            << 2*bufferSize*nIDs *sizeof(ID_t)/1024.<< " kB \n";
  BondHistoryID1 = new MirroredVector< ID_t >( 2*bufferSize*nIDs, mStream ); //essentially the ids of the first monomer 
  BondHistoryID2 = new MirroredVector< ID_t >( 2*bufferSize*nIDs, mStream ); //essentially the ids of the second monomer
}
void Tracker::increaseCounter()
{
  counter++;
}


void Tracker::dumpReactions()
{
  BaseClass::setBufferSize(bufferSize);
  BondHistoryID1->popAsync();
  BondHistoryID2->popAsync();
  CUDA_ERROR( hipStreamSynchronize( mStream ) );
  std::cout <<"counter= " << counter << " nIDs="<<nIDs << std::endl;
  for (uint32_t j=0 ; j < counter ; j ++ ) {
  uint32_t currentAge(age[j]);
    for(uint32_t i =0 ; i < nIDs; i ++)
    {
      auto index(i+nIDs*j);
      auto Mon1(BondHistoryID1->host[index]);
      auto Mon2(BondHistoryID2->host[index]);
      
      if( Mon2  > 0 )
      {
	std::vector<uint32_t> vec;
	vec.push_back(currentAge); //time 
	vec.push_back( Mon2 & 1 ); // either 0 or 1 for remove or add 
	Mon2 = (Mon2 >> 1) -1;
	vec.push_back(std::min(Mon1,Mon2)); 
	vec.push_back(std::max(Mon1,Mon2));
	BaseClass::addConnection(vec);
	BondHistoryID1->host[index]=0;
	BondHistoryID2->host[index]=0;
      }
    }
  }
  BaseClass::dumpReactions();
  counter=0;
  age.resize(0);
  BondHistoryID1->push();
  BondHistoryID2->push();
  
}