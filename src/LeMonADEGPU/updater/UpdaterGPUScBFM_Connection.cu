#include "hip/hip_runtime.h"


/*
 * UpdaterGPUScBFM_Connection.cu
 *
 *  Created on: 27.06.2019
 *      Authors: Toni Mueller
 */

#include <LeMonADEGPU/updater/UpdaterGPUScBFM_Connection.h>
// #include <LeMonADEGPU/updater/UpdaterGPUScBFM_AB_Type.h>
#include <LeMonADEGPU/utility/cudacommon.hpp>
#include <LeMonADEGPU/core/Method.h>
#include <LeMonADEGPU/utility/DeleteMirroredObject.h>
#include <hip/hip_runtime_api.h>              // hipProfilerStop
#include <LeMonADEGPU/utility/AutomaticThreadChooser.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <extern/Fundamental/BitsCompileTime.hpp>
#include <LeMonADEGPU/utility/cudacommon.hpp>

#include <LeMonADEGPU/utility/SelectiveLogger.hpp>
#include <LeMonADEGPU/utility/graphColoring.tpp>
#include <LeMonADEGPU/core/rngs/Saru.h>
#include <LeMonADEGPU/core/MonomerEdges.h>
#include <LeMonADEGPU/core/constants.cuh>
#include <LeMonADEGPU/feature/BoxCheck.h>
#include <LeMonADEGPU/core/Method.h>

#include <LeMonADEGPU/utility/DeleteMirroredObject.h>
#include <LeMonADEGPU/core/BondVectorSet.h>


using T_Flags            = UpdaterGPUScBFM_Connection< uint8_t >::T_Flags      ;
__device__ __constant__ uint32_t dcCrossLinkMaxNumLinks     ;  // functionality of cross links 
__device__ __constant__ uint32_t dcChainMaxNumLinks =  2    ;  // functionality of chain ends 
/**
 * @brief convinience function to print the box dimensions for the device constants 
 */
__global__ void CheckBoxDimensions()
{
printf("KernelCheckBoxDimensions: %d %d %d %d %d %d  %d %d \n",dcBoxX,dcBoxY, dcBoxZ,dcBoxXM1, dcBoxYM1,dcBoxZM1, dcBoxXLog2, dcBoxXYLog2 );
}

/**
 * @brief writes the ID of the chain ends on the lattice
 * @details The ID start at 1 and are shifted by and offset which is given
 * 	    by the previous species of monomers. 
 */
template< typename T_UCoordinateCuda >
__global__ void kernelUpdateReactiveLattice
(
    typename CudaVec4< T_UCoordinateCuda >::value_type
                        const * const __restrict__ dpPolymerSystem  ,
    uint32_t            const                      iOffset          ,
    T_Id                      * const __restrict__ dpReactiveLattice,
    T_Id                        const              nMonomers        ,
    Method                      const              met 
)
{
    for ( T_Id iMonomer = blockIdx.x * blockDim.x + threadIdx.x;
          iMonomer < nMonomers; iMonomer += gridDim.x * blockDim.x )
    {
        auto const r0 = dpPolymerSystem[ iOffset + iMonomer ];
	auto const Vector(met.getCurve().linearizeBoxVectorIndex( r0.x, r0.y, r0.z ) );
// 	dpReactiveLattice[ Vector ] = (iOffset+ iMonomer+1 );
	dpReactiveLattice[ Vector ] = ( iMonomer+1 );
    }
}
 /**
  * @brief convinience function to update the lattice occupation. 
  * @details We introduce such functions because then they can be used latter on from inheritate classes..
  */
template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::launch_initializeReactiveLattice(
  const size_t nBlocks , const size_t nThreads, const T_Id iSpecies )
{
  mLog ( "Check" ) <<"Start filling lattice with ones:  \n" ;
  mLatticeIds->memset(0,0);

  mLog ( "Check" ) << "launch_initializeReactiveLattice:: iSpecies = " << iSpecies <<"\n"
		  << "launch_initializeReactiveLattice:: mviSubGroupOffsets[ iSpecies ] = "<< mviSubGroupOffsets[ iSpecies ]<<"\n"
		  << "launch_initializeReactiveLattice:: mnElementsInGroup[ iSpecies ] = "<< mnElementsInGroup[ iSpecies ]<<"\n";
  if ( false ){ //fill in cpu 
    mPolymerSystemSorted->pop();
    for (T_Id i =0; i < mnElementsInGroup[ iSpecies ]; i++)
    {
      auto const iMonomer(i+mviSubGroupOffsets[ iSpecies ]);
      auto const r(mPolymerSystemSorted->host[iMonomer]); 
      auto const Vector(met.getCurve().linearizeBoxVectorIndex(r.x,r.y,r.z));
      mLatticeIds->host[Vector]= iMonomer+1;
    }
    mLatticeIds->push(0);
    hipStreamSynchronize( mStream );
  }else {
      kernelUpdateReactiveLattice<T_UCoordinateCuda><<<nBlocks,nThreads,0,mStream>>>(
	  mPolymerSystemSorted->gpu     ,            
	  mviSubGroupOffsets[ iSpecies ], 
	  mLatticeIds->gpu              ,
	  mnElementsInGroup[ iSpecies ] ,                        
	  met
      );
  }
}
/**
 * @brief Counts the number of occupied lattice sites.
 */
template< typename T_UCoordinateCuda  >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::checkLatticeOccupation()  
{
  mLatticeIds->pop(0);
  uint32_t countLatticeEntries(0);
  std::cout <<  "BoxDim=("<<mBoxX<<","<<mBoxY<<","<<mBoxZ<<")"<<std::endl;
  for(T_Id x=0; x< mBoxX; x++ )
    for(T_Id y=0; y< mBoxY; y++ )
      for(T_Id z=0; z< mBoxX; z++ )
	if(mLatticeIds->host[met.getCurve().linearizeBoxVectorIndex(x,y,z)] > 0 )
	  countLatticeEntries++;
    mLog( "Info" )
        << "checkLatticeOccupation: \n"
	<< "nReactiveMonomersChains = " << nReactiveMonomersChains << "\n"
	<< "countLatticeEntries     = " << countLatticeEntries << "\n";
}

/**
 * @brief checks the lattice for possible neighbors
 * @details We randomly choose a direction and look on the lattice for a possibel
 * 	    new partner. In this case the lattice entry is greater one. 
 */
template< typename T_UCoordinateCuda >
__global__ void kernelCheckConnection
(
    typename CudaVec4< T_UCoordinateCuda >::value_type
                const * const __restrict__ dpPolymerSystem          ,
    uint32_t            const              iOffset                  ,
    T_Id              * const              dLatticeIds              ,
    T_Id              * const              dpFlag                   ,
    uint8_t     const * const              dpNeighborsSizesCrossLink,
    uint8_t     const * const              dpNeighborsSizesChain    ,
    T_Id                const              nMonomers                ,
    uint64_t            const              rSeed                    ,
    uint64_t            const              rGlobalIteration         ,
    Method              const              met
){
    uint32_t rn;
    int iGrid = 0;
    for ( uint32_t iMonomer = blockIdx.x * blockDim.x + threadIdx.x;
          iMonomer < nMonomers; iMonomer += gridDim.x * blockDim.x, ++iGrid )
    {
        auto const r0 = dpPolymerSystem[ iOffset + iMonomer ];
	if ( dcCrossLinkMaxNumLinks == dpNeighborsSizesCrossLink[ iMonomer ] ) continue; //already max number of connections for the crosslinker
        if ( iGrid % 1 == 0 ) //for what is this  
        {
	  Saru rng(rGlobalIteration,iMonomer,rSeed);
	  rn =rng.rng32();
        }
        int direction = rn % 6;
	/* select random direction. Do this with bitmasking instead of lookup ??? */
	
        typename CudaVec4< T_UCoordinateCuda >::value_type const r1 = {
            T_UCoordinateCuda( r0.x + DXTable2_d[ direction ] ),
            T_UCoordinateCuda( r0.y + DYTable2_d[ direction ] ),
            T_UCoordinateCuda( r0.z + DZTable2_d[ direction ] ) };
	
// 	auto const PartnerlatticeEntry = tex1Dfetch<T_Id>(texLatticeIds, met.getCurve().linearizeBoxVectorIndex(r1.x,r1.y,r1.z ) );
	auto const PartnerlatticeEntry = dLatticeIds[met.getCurve().linearizeBoxVectorIndex(r1.x,r1.y,r1.z )];
	printf("ng=%d max_X=%d l=%d, (%d,%d,%d), (%d,%d,%d), lvec=%d 2*DXTable2_d[0]=%d\n", dpNeighborsSizesCrossLink[ iMonomer ], dcCrossLinkMaxNumLinks, PartnerlatticeEntry, r1.x,r1.y,r1.z ,r0.x,r0.y,r0.z, met.getCurve().linearizeBoxVectorIndex(r1.x,r1.y,r1.z ) , DXTable2_d[0]);
	//Partner Id start at 1!!!
	if ( PartnerlatticeEntry == 0 ) continue; //is not reactive for 0  or cross link (do not allow connections betweeen cross links)
	if ( dcChainMaxNumLinks == dpNeighborsSizesChain[ PartnerlatticeEntry -1 ] ) continue; //already max number of connections for the chain
        dpFlag[ iMonomer + 1 ] = PartnerlatticeEntry ; 
    }
}
template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::launch_CheckConnection(
  const size_t nBlocks, const size_t nThreads, 
  const size_t iSpeciesCrossLink, const size_t iSpeciesChain,const uint64_t seed)
{
  mLog( "Stats" ) << "Start kernel kernelCheckConnection: \n";
//   uint32_t FoundPotentialPartner(0);
//   mPolymerSystemSorted->pop(0);
//   for(uint32_t i =0; i <  mnElementsInGroup[ iSpeciesCrossLink ]; i++){
//     auto const  r0(mPolymerSystemSorted->host[i+mviSubGroupOffsets[ iSpeciesCrossLink ]]);
//     typename CudaVec4< T_UCoordinateCuda >::value_type const r1 = {
//             T_UCoordinateCuda( r0.x + 2 ), T_UCoordinateCuda( r0.y ), T_UCoordinateCuda( r0.z ) };
//     auto LatticeEntry(mLatticeIds->host[met.getCurve().linearizeBoxVectorIndex(r1.x,r1.y,r1.z )]);
//     if (LatticeEntry != 0 )
//     FoundPotentialPartner++;
//   }
//   std::cout << "found " << FoundPotentialPartner<<std::endl;
  kernelCheckConnection< T_UCoordinateCuda > 
  <<<nBlocks, nThreads, 0, mStream>>>(                
      mPolymerSystemSorted->gpu,       
      mviSubGroupOffsets[ iSpeciesCrossLink ], 
      mLatticeIds->gpu,
      mCrossLinkFlags,
      mNeighborsSortedSizes->gpu + mviSubGroupOffsets[ iSpeciesCrossLink ], 
      mNeighborsSortedSizes->gpu + mviSubGroupOffsets[ iSpeciesChain ], 
      mnElementsInGroup[ iSpeciesCrossLink ],                       
      seed, 
      mGlobalIterator,                                         
      met
  );
  mGlobalIterator++;
  mLog( "Stats" ) << "Start kernel kernelCheckConnection.done \n";
}



template< typename T_UCoordinateCuda >
__global__ void kernelApplyConnection
(
    T_Id              * const              mCrossLinkFlags         ,
    T_Id              * const              mCrossLinkIDS           ,
    T_Id                const              flagArraySize           ,               
    T_Id              * const              dpNeighborsMonomer      ,
    T_Id              * const              dpNeighborsPartner      ,
    uint32_t            const              rNeighborsPitchElementsMonomer ,
    uint32_t            const              rNeighborsPitchElementsPartner ,
    uint8_t           * const              dpNeighborsSizesMonomer ,
    uint8_t           * const              dpNeighborsSizesPartner 
    
){
    for ( auto i = blockIdx.x * blockDim.x + threadIdx.x;
          i < flagArraySize; i += gridDim.x * blockDim.x )
    {
      auto iPartner(mCrossLinkFlags[i]);
      auto iMonomer(mCrossLinkIDS[i]);
      if (iPartner == 0 || iMonomer == 0 ) 
      {
	mCrossLinkFlags[i]=0;
	mCrossLinkIDS[i]=0;
	continue; //no Partner found -> go to next Crosslink in the grid 
      }
      iPartner--;
      iMonomer--;
//       printf("Connect monomers: %d with %d \n", iMonomer, iPartner ); 
      dpNeighborsMonomer[ dpNeighborsSizesMonomer[ iMonomer ] * rNeighborsPitchElementsMonomer + iMonomer ] = iPartner; 
      dpNeighborsPartner[ dpNeighborsSizesPartner[ iPartner ] * rNeighborsPitchElementsPartner + iPartner ] = iMonomer; 
      dpNeighborsSizesMonomer[ iMonomer ]++;
      dpNeighborsSizesPartner[ iPartner ]++; 
      printf("Connect monomers: %d with %d , %d ,%d ,%d ,%d \n", iMonomer, iPartner, 
	     dpNeighborsMonomer[ (dpNeighborsSizesMonomer[ iMonomer ]-1) * rNeighborsPitchElementsMonomer + iMonomer ], 
	     dpNeighborsPartner[ (dpNeighborsSizesPartner[ iPartner ]-1) * rNeighborsPitchElementsPartner + iPartner ],
	     dpNeighborsSizesMonomer[ iMonomer ],
	     dpNeighborsSizesPartner[ iPartner ]
	    ); 
      mCrossLinkFlags[i]=0;
      mCrossLinkIDS[i]=0;
    }
}
#include <LeMonADEGPU/core/kernelConnection.h>
template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::launch_ApplyConnection(
  const size_t nBlocks , const size_t   nThreads, 
  const size_t MonomerSpecies,
  const size_t PartnerSpecies
)
{
  //reset vectors 
  thrust::sequence(thrust::device, mCrossLinkIDS, mCrossLinkIDS+flagArraySize,1 );
  Connection connection(flagArraySize);
  connection.resetMultipleIDs(mCrossLinkIDS,mCrossLinkFlags);
  kernelApplyConnection<T_UCoordinateCuda><<<nBlocks,nThreads,0,mStream>>>(
    mCrossLinkFlags,
    mCrossLinkIDS,
    flagArraySize, 
    mNeighborsSorted->gpu + mNeighborsSortedInfo.getMatrixOffsetElements( MonomerSpecies ), 
    mNeighborsSorted->gpu + mNeighborsSortedInfo.getMatrixOffsetElements( PartnerSpecies ), 
    mNeighborsSortedInfo.getMatrixPitchElements( MonomerSpecies ),
    mNeighborsSortedInfo.getMatrixPitchElements( PartnerSpecies ),       
    mNeighborsSortedSizes->gpu + mviSubGroupOffsets[ MonomerSpecies ],
    mNeighborsSortedSizes->gpu + mviSubGroupOffsets[ PartnerSpecies ]
  );

  
}



template< typename T_UCoordinateCuda > 
UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::UpdaterGPUScBFM_Connection():
BaseClass()                         , 
mLatticeIds                 ( NULL ),
mCrossLinkFlags             ( NULL ),
mCrossLinkIDS               ( NULL ),
nReactiveMonomers           ( 0    ),
nReactiveMonomersChains     ( 0    ),
nReactiveMonomersCrossLinks ( 0    ),
crosslinkFunctionality      ( 0    )
{
    /**
     * Log control.
     * Note that "Check" controls not the output, but the actualy checks
     * If a checks needs to always be done, then do that check and declare
     * the output as "Info" log level
     */
    mLog.file( __FILENAME__ );
    mLog.deactivate( "Check"     );
    mLog.deactivate( "Error"     );
    mLog.deactivate( "Info"      );
    mLog.deactivate( "Stats"     );
    mLog.deactivate( "Warning"   );
};
template< typename T_UCoordinateCuda > 
void UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::destruct(){
      
    DeleteMirroredObject deletePointer;
    deletePointer( mLatticeIds       , "mLatticeIds"        );
    //Why do i get a device pointer error ?!
//     CUDA_ERROR(hipFree(mCrossLinkFlags));
//     CUDA_ERROR(hipFree(mCrossLinkIDS));
    if ( deletePointer.nBytesFreed > 0 )
    {
        mLog( "Info" )
            << "Freed a total of "
            << prettyPrintBytes( deletePointer.nBytesFreed )
            << " on GPU and host RAM.\n";
    }
}
template< typename T_UCoordinateCuda > 
UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::~UpdaterGPUScBFM_Connection()
{
  this->destruct();    
}

template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::cleanup()
{
    BaseClass::destruct();
    this->destruct();    
    hipDeviceSynchronize();
    hipProfilerStop();
    
}

template < typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::initialize()
{
  BaseClass::setAutoColoring(false);
  mLog( "Info" )<< "Start manual coloring of the graph...\n" ;
  //do manual coloring 
  for ( auto i = 0; i < mnAllMonomers ; i++)
  {
    T_Id color(( i % 2)==0 ? 2 :3);
    mGroupIds.push_back(color ); 
  }

  for (auto i = 0; i < nReactiveMonomers; i++)
  {
    mGroupIds[mNewToOldReactiveID[i]] = (mMonomerReactivity[i].maxNumLinks == 2 ) ? 1 : 0 ;
    if (i <20 ) 
      mLog( "Info" )<< "mGroups[" << mNewToOldReactiveID[i] << "]= "<< mGroupIds[mNewToOldReactiveID[i]] <<"\n" ;
  }
  mLog( "Info" )<< "Start manual coloring of the graph...done\n" ;
  mLog( "Info" )<< "Initialize baseclass \n" ;
  BaseClass::initialize();

  
  mLog( "Info" )<< "Allocate memory on gpu. \n" ;
  mLog( "Info" )<<"Cross link functionality is "<< crosslinkFunctionality << "\n";
  CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcCrossLinkMaxNumLinks), &crosslinkFunctionality, sizeof( crosslinkFunctionality ) ) );
//   CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcChainMaxNumLinks), crosslinkFunctionality, sizeof( crosslinkFunctionality ) ) );
  flagArraySize = (4*ceil(nReactiveMonomersCrossLinks*1.0/4.) );
  CUDA_ERROR(hipMalloc((void **) &mCrossLinkIDS, sizeof(T_Id)*flagArraySize));
  CUDA_ERROR(hipMalloc((void **) &mCrossLinkFlags, sizeof(T_Id)*flagArraySize));
  mLog( "Info" )<< "Allocate memory on gpu.done. \n" ;
      

  { decltype( dcBoxX      ) x = mBoxX     ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxX     ), &x, sizeof(x) ) ); }
  { decltype( dcBoxY      ) x = mBoxY     ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxY     ), &x, sizeof(x) ) ); }
  { decltype( dcBoxZ      ) x = mBoxZ     ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxZ     ), &x, sizeof(x) ) ); }
  { decltype( dcBoxXM1    ) x = mBoxXM1   ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXM1   ), &x, sizeof(x) ) ); }
  { decltype( dcBoxYM1    ) x = mBoxYM1   ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxYM1   ), &x, sizeof(x) ) ); }
  { decltype( dcBoxZM1    ) x = mBoxZM1   ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxZM1   ), &x, sizeof(x) ) ); }
  { decltype( dcBoxXLog2  ) x = mBoxXLog2 ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXLog2 ), &x, sizeof(x) ) ); }
  { decltype( dcBoxXYLog2 ) x = mBoxXYLog2; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXYLog2), &x, sizeof(x) ) ); }
  uint32_t tmp_DXTable2[6] = { 0u-2u,2,  0,0,  0,0 };
  uint32_t tmp_DYTable2[6] = {  0,0, 0u-2u,2,  0,0 };
  uint32_t tmp_DZTable2[6] = {  0,0,  0,0, 0u-2u,2 };
  CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( DXTable2_d), tmp_DXTable2, sizeof( tmp_DXTable2 ) ) );
  CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( DYTable2_d), tmp_DYTable2, sizeof( tmp_DXTable2 ) ) );
  CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( DZTable2_d), tmp_DZTable2, sizeof( tmp_DXTable2 ) ) );
  if (mLog( "Info" ).isActive()){
    mLog( "Info" )<< "Check box dimensions \n" ;
    CheckBoxDimensions<<<1,1,0,mStream>>>();
    CUDA_ERROR( hipStreamSynchronize( mStream ) ); // finish e.g. initializations
  }
  mLog( "Info" )<< "Initialize baseclass.done. \n" ;
  CrossLinkSpecies = 0; 
  ChainEndSpecies  = 1; 
  initializeReactiveLattice();
  mLog( "Info" )<< "Initialize lattice.done. \n" ;

}

template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::setNrOfReactiveMonomers( T_Id nReactiveMonomers_ , T_Id nReactiveMonomersCrossLinks_, T_Id nReactiveMonomersChains_ )
{
    if ( nReactiveMonomers != 0 || nReactiveMonomersCrossLinks != 0 || nReactiveMonomersChains != 0 )
    {
        std::stringstream msg;
        msg << "[" << __FILENAME__ << "::setNrOfReactiveMonomers] "
            << "Number of nReactiveMonomers already set to           " << nReactiveMonomers << "!\n"
	    << "Number of nReactiveMonomersChains already set to     " << nReactiveMonomersChains << "!\n"
	    << "Number of nReactiveMonomersCrossLinks already set to " << nReactiveMonomersCrossLinks << "!\n";
        throw std::runtime_error( msg.str() );
    }
    nReactiveMonomers           = nReactiveMonomers_;
    nReactiveMonomersChains     = nReactiveMonomersChains_;
    nReactiveMonomersCrossLinks = nReactiveMonomersCrossLinks_;
    mLog( "Info" )
	  << "Nr of reactive monomers   "<< nReactiveMonomers <<"\n" 
	  << "Nr of reactive crosslinks "<< nReactiveMonomersCrossLinks <<"\n" 
	  << "Nr of reactive chain ends "<< nReactiveMonomersChains <<"\n" ;
    
};

template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::setReactiveGroup(T_Id monID_, bool reactivity_, T_MaxNumLinks maxNumLinks_){

  
  //fill  mMonomerReactivity
  if (reactivity_ ){
    mNewToOldReactiveID.push_back(monID_);
    D_MonomerReactivity monReact;
    monReact.reactivity=reactivity_;
    monReact.maxNumLinks=maxNumLinks_;
    mMonomerReactivity.push_back(monReact);
    if(maxNumLinks_ > crosslinkFunctionality) crosslinkFunctionality = maxNumLinks_;
  }
  
}

template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection<T_UCoordinateCuda>::initializeReactiveLattice()
{
 if ( mLatticeIds != NULL )
    {
        std::stringstream msg;
        msg << "[" << __FILENAME__ << "::initializeReactiveLattice] "
            << "Initialize was already called and may not be called again "
            << "until cleanup was called!";
        mLog( "Error" ) << msg.str();
        throw std::runtime_error( msg.str() );
    }
    mLog( "Info" ) << "Allocate memory for lattice \n";  
    size_t nBytesLatticeTmp = mBoxX * mBoxY * mBoxZ * sizeof(T_Id);
    mLatticeIds  = new MirroredTexture< T_Id >( nBytesLatticeTmp, mStream );

//     /* populate latticeOut with monomers from mPolymerSystem */
//     std::memset( mLatticeIds->host, 0, mLatticeIds->nBytes );
//     for ( T_Id i = 0; i < nReactiveMonomers; ++i )
//     {
//       
//     }
// 	auto iMonomer(mNewToOldReactiveID[i]);
// 	T_Id latticeEntry(0);
// 	//write only chain ends on the lattice (starting at 1)!!!
// 	if (mMonomerReactivity->host[iMonomer].reactivity  == 1 && // this statement should always be true, because this objects only contains reactive monomers...
// 	    mMonomerReactivity->host[iMonomer].maxNumLinks == 2 
// 	)
// 	{
// 	 //write the new id monomers on the lattice,because they are used for the neighbor information... 
// 	 latticeEntry = miToiNew->host[iMonomer] +1; 
// 	}
// 	
//         mLatticeIds->host[ met.getCurve().linearizeBoxVectorIndex(
//             mPolymerSystem->host[ iMonomer ].x,
//             mPolymerSystem->host[ iMonomer ].y,
//             mPolymerSystem->host[ iMonomer ].z
//         ) ] = latticeEntry;
//     }
//     mLatticeIds->pushAsync();d
    auto const nThreads = 128; 
    auto const nBlocks  = ceilDiv( mnElementsInGroup[ ChainEndSpecies ], nThreads );
    mLog( "Info" )
        << "Start kernel for initialization of the reactive lattice. " 
	<< "Using nThreads: " << nThreads << "\n"
	<< "      nBlocks : " << nBlocks  << "\n"
	<< "Nr of chain elements: " << mnElementsInGroup[ ChainEndSpecies ] 
	<< "\n";
    met.modifyCurve().setMode(2);
    launch_initializeReactiveLattice(nBlocks, nThreads, ChainEndSpecies);
    mLog( "Info" )
        << "Filling Rate of reactive monomers: " << nReactiveMonomers << " "
        << "(=" << nReactiveMonomers / 1024 << "*1024+" << nReactiveMonomers % 1024 << ") "
        << "particles in a (" << mBoxX << "," << mBoxY << "," << mBoxZ << ") box "
        << "=> " << 100. * nReactiveMonomers / ( mBoxX * mBoxY * mBoxZ ) << "%\n"
        << "Note: densest packing is: 25% -> in this case it might be more reasonable to actually iterate over the spaces where particles can move to, keeping track of them instead of iterating over the particles\n";
    checkLatticeOccupation();
    met.modifyCurve().setMode(0);
    
}



template< typename T_UCoordinateCuda  >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::runSimulationOnGPU
(
    uint32_t const nMonteCarloSteps
)
{
    std::clock_t const t0 = std::clock();

    CUDA_ERROR( hipStreamSynchronize( mStream ) ); // finish e.g. initializations
    CUDA_ERROR( hipMemcpy( mPolymerSystemSortedOld->gpu, mPolymerSystemSorted->gpu, mPolymerSystemSortedOld->nBytes, hipMemcpyDeviceToDevice ) );
    auto const nSpecies = mnElementsInGroup.size();
    AutomaticThreadChooser chooseThreads(nSpecies);
    chooseThreads.initialize(mCudaProps);
    std::vector< uint64_t > nSpeciesChosen( nSpecies ,0 );

    /* run simulation */
    for ( uint32_t iStep = 0; iStep < nMonteCarloSteps; ++iStep, ++mAge )
    {
        if ( mUsePeriodicMonomerSorting && ( mAge % mnStepsBetweenSortings == 0 ) )
        {
            mLog( "Info" ) << "Resorting at age / step " << mAge << "\n";
            doSpatialSorting();
        }
        if ( useOverflowChecks )
        {
            /**
             * for uint8_t we have to check for overflows every 127 steps, as
             * for 128 steps we couldn't say whether it actually moved 128 steps
             * or whether it moved 128 steps in the other direction and was wrapped
             * to be equal to the hypothetical monomer above
             */
            auto constexpr boxSizeCudaType = 1ll << ( sizeof( T_UCoordinateCuda ) * CHAR_BIT );
            auto constexpr nStepsBetweenOverflowChecks = boxSizeCudaType / 2 - 1;
            if ( iStep != 0 && iStep % nStepsBetweenOverflowChecks == 0 )
            {
                findAndRemoveOverflows( false );
                CUDA_ERROR( hipMemcpyAsync( mPolymerSystemSortedOld->gpu,
                    mPolymerSystemSorted->gpu, mPolymerSystemSortedOld->nBytes,
                    hipMemcpyDeviceToDevice, mStream ) );
            }
        }
        /* one Monte-Carlo step:
         *  - tries to move on average all particles one time
         *  - each particle could be touched, not just one group */
        for ( uint32_t iSubStep = 0; iSubStep < nSpecies; ++iSubStep ) {
            auto const iStepTotal = iStep * nSpecies + iSubStep;
            auto  iOffsetLatticeTmp = ( iStepTotal % mnLatticeTmpBuffers )
            * ( mBoxX * mBoxY * mBoxZ * sizeof( mLatticeTmp->gpu[0] ));
            if (met.getPacking().getBitPackingOn()) 
                iOffsetLatticeTmp /= CHAR_BIT;
            auto texLatticeTmp = mvtLatticeTmp[ iStepTotal % mnLatticeTmpBuffers ];

            if (met.getPacking().getNBufferedTmpLatticeOn()) {
                    iOffsetLatticeTmp = 0u;
                    texLatticeTmp = mLatticeTmp->texture;
            }
            /* randomly choose which monomer group to advance */
            auto const iSpecies = randomNumbers.r250_rand32() % nSpecies;
            auto const seed     = randomNumbers.r250_rand32();
            auto const nThreads = chooseThreads.getBestThread(iSpecies);
            auto const nBlocks  = ceilDiv( mnElementsInGroup[ iSpecies ], nThreads );
            auto const useCudaMemset = chooseThreads.useCudaMemset(iSpecies);
            chooseThreads.addRecord(iSpecies, mStream);

            nSpeciesChosen[ iSpecies ] += 1;

            launch_CheckSpecies(nBlocks, nThreads, iSpecies, iOffsetLatticeTmp, seed);

            if ( useCudaMemset )
                launch_PerformSpeciesAndApply(nBlocks, nThreads, iSpecies, texLatticeTmp);
            else
                launch_PerformSpecies(nBlocks,nThreads,iSpecies,texLatticeTmp);

            if ( useCudaMemset ){
                if(met.getPacking().getNBufferedTmpLatticeOn()){
                    /* we only need to delete when buffers will wrap around and
                        * on the last loop, so that on next runSimulationOnGPU
                        * call mLatticeTmp is clean */
                    if ( ( iStepTotal % mnLatticeTmpBuffers == 0 ) ||
                        ( iStep == nMonteCarloSteps-1 && iSubStep == nSpecies-1 ) )
                    {
                        hipMemsetAsync( (void*) mLatticeTmp->gpu, 0, mLatticeTmp->nBytes, mStream );
                    }
                }else
                    mLatticeTmp->memsetAsync(0);
            }
            else
                launch_ZeroArraySpecies(nBlocks,nThreads,iSpecies);
	    //updates the reactiveLattice 
	    //could be incoporated into the launch_PerformSpecies kernel ...
            chooseThreads.analyze(iSpecies,mStream);
        } // iSubstep
//         met.modifyCurve().setMode(2);
// 	checkLatticeOccupation();
// 	auto const nThreads = chooseThreads.getBestThread(ChainEndSpecies);
// 	auto const nBlocks  = ceilDiv( mnElementsInGroup[ ChainEndSpecies ], nThreads );
//         launch_initializeReactiveLattice( nBlocks, nThreads, ChainEndSpecies);
// 	checkLatticeOccupation();
// 	auto const nThreads_c = 128;
// 	auto const nBlocks_c  = ceilDiv( nReactiveMonomersCrossLinks, nThreads_c );
// 	auto const seed     = randomNumbers.r250_rand32();
//         launch_CheckConnection(nBlocks_c,nThreads_c,CrossLinkSpecies, ChainEndSpecies,seed);
// 	launch_ApplyConnection(nBlocks_c,nThreads_c,CrossLinkSpecies, ChainEndSpecies);
// 	met.modifyCurve().setMode(0);
    } // iStep
    
    doCopyBack();
    checkSystem(); // no-op if "Check"-level deactivated
    std::clock_t const t1 = std::clock();
    double const dt = float(t1-t0) / CLOCKS_PER_SEC;
    mLog( "Info" )
    << "run time (GPU): " << nMonteCarloSteps << "\n"
    << "mcs = " << nMonteCarloSteps  << "  speed [performed monomer try and move/s] = MCS*N/t: "
    << nMonteCarloSteps * ( mnAllMonomers / dt )  << "     runtime[s]:" << dt << "\n";
}

template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::doCopyBack()
{
    mLog( "Info" ) << "UpdaterGPUScBFM_AB_Type< T_UCoordinateCuda >::doCopyBackConnectivity() \n";
    doCopyBackMonomerPositions();
    mLog( "Info" ) << "UpdaterGPUScBFM_AB_Type< T_UCoordinateCuda >::doCopyBackConnectivity() \n";
    doCopyBackConnectivity();
  
}


template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::checkBonds() const
{ 
    /**
     * Check bonds i.e. that |dx|<=3 and whether it is allowed by the given
     * bond set
     */
    for ( T_Id i = 0; i < mnAllMonomers; ++i )
    for ( unsigned iNeighbor = 0; iNeighbor < mNeighbors->host[i].size; ++iNeighbor )
    {
        /* calculate the bond vector between the neighbor and this particle
         * neighbor - particle = ( dx, dy, dz ) */
        auto const neighbor = mPolymerSystem->host[ mNeighbors->host[i].neighborIds[ iNeighbor ] ];
        auto dx = (int) neighbor.x - (int) mPolymerSystem->host[i].x;
        auto dy = (int) neighbor.y - (int) mPolymerSystem->host[i].y;
        auto dz = (int) neighbor.z - (int) mPolymerSystem->host[i].z;
        /* with this uncommented, we can ignore if a monomer jumps over the
         * whole box range or T_UCoordinateCuda range */
        dx %= mBoxX; if ( dx < -int( mBoxX )/ 2 ) dx += mBoxX; if ( dx > (int) mBoxX / 2 ) dx -= mBoxX;
        dy %= mBoxY; if ( dy < -int( mBoxY )/ 2 ) dy += mBoxY; if ( dy > (int) mBoxY / 2 ) dy -= mBoxY;
        dz %= mBoxZ; if ( dz < -int( mBoxZ )/ 2 ) dz += mBoxZ; if ( dz > (int) mBoxZ / 2 ) dz -= mBoxZ;
        int erroneousAxis = -1;
        if ( ! ( -3 <= dx && dx <= 3 ) ) erroneousAxis = 0;
        if ( ! ( -3 <= dy && dy <= 3 ) ) erroneousAxis = 1;
        if ( ! ( -3 <= dz && dz <= 3 ) ) erroneousAxis = 2;
        if ( erroneousAxis >= 0 || checkBondVector( dx, dy, dz )  )
        {
            std::stringstream msg;
            msg << "[" << __FILENAME__ << "::checkSystem] ";
            if ( erroneousAxis > 0 )
                msg << "Invalid " << char( 'X' + erroneousAxis ) << "-Bond: ";
            if ( checkBondVector( dx, dy, dz ) )
                msg << "This particular bond is forbidden: ";
            msg << "(" << dx << "," << dy<< "," << dz << ") between monomer "
                << i << " at (" << mPolymerSystem->host[i].x << ","
                                << mPolymerSystem->host[i].y << ","
                                << mPolymerSystem->host[i].z << ") and monomer "
                << mNeighbors->host[i].neighborIds[ iNeighbor ] << " at ("
                << neighbor.x << "," << neighbor.y << "," << neighbor.z << ")"
                << std::endl;
             throw std::runtime_error( msg.str() );
        }
    } 
}

template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_Connection< T_UCoordinateCuda >::checkSystem() const
{
    if ( ! mLog.isActive( "Check" ) )
        return;
    BaseClass::checkLatticeOccupation();
    checkBonds();
}



template class UpdaterGPUScBFM_Connection< uint8_t  >;
template class UpdaterGPUScBFM_Connection< uint16_t >;
template class UpdaterGPUScBFM_Connection< uint32_t >;
template class UpdaterGPUScBFM_Connection<  int16_t >;
template class UpdaterGPUScBFM_Connection<  int32_t >;


