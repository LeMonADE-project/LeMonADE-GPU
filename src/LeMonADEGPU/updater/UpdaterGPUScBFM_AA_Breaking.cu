#include "hip/hip_runtime.h"


/*
 * UpdaterGPUScBFM_AA_Breaking.cu
 *
 *  Created on: 27.06.2019
 *      Authors: Toni Mueller
 */

#include <LeMonADEGPU/updater/UpdaterGPUScBFM_AA_Breaking.h>
// #include <LeMonADEGPU/updater/UpdaterGPUScBFM_AB_Type.h>
#include <LeMonADEGPU/utility/cudacommon.hpp>
#include <LeMonADEGPU/core/Method.h>
#include <LeMonADEGPU/utility/DeleteMirroredObject.h>
#include <hip/hip_runtime_api.h>              // hipProfilerStop
#include <LeMonADEGPU/utility/AutomaticThreadChooser.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <extern/Fundamental/BitsCompileTime.hpp>

#include <LeMonADEGPU/utility/SelectiveLogger.hpp>
#include <LeMonADEGPU/utility/graphColoring.tpp>
#include <LeMonADEGPU/core/rngs/Saru.h>
#include <LeMonADEGPU/core/MonomerEdges.h>
#include <LeMonADEGPU/core/constants.cuh>
#include <LeMonADEGPU/feature/BoxCheck.h>
#include <LeMonADEGPU/core/Method.h>

#include <LeMonADEGPU/utility/DeleteMirroredObject.h>
#include <LeMonADEGPU/core/BondVectorSet.h>
#include <LeMonADEGPU/core/kernelConnection.h>
#include <LeMonADEGPU/utility/GPUConnectionTracker.h>
#include <math.h>

__device__ __constant__ double dcBreakingProbability     ;  // functionality of cross links 
__global__ void kernel_BreakConnections
(
    uint8_t           * const              dpNeighborsSizes               ,
    T_Id              * const              dpNeighbors                    ,
    uint32_t            const              rNeighborsPitchElements        ,
    uint32_t 		const 		   iOffset                        ,
    T_Id                const              nMonomers                      ,
    uint64_t            const              rSeed                          ,
    uint64_t            const              rGlobalIteration               ,
    uint8_t           * const              texAllowedToMove	          ,
    T_Id              * const              dBreaks
)
{
    double rn;
    int iGrid;
    for ( uint32_t iMonomer = blockIdx.x * blockDim.x + threadIdx.x;
          iMonomer < nMonomers; iMonomer += gridDim.x * blockDim.x, iGrid++)
    {   
      if ( texAllowedToMove[iMonomer] == 1 )
      {
	if (iGrid %1 ==0 ){
	  Saru rng(rGlobalIteration,iMonomer,rSeed);
	  rn =rng.rng_d(); 
	  if (rn < dcBreakingProbability)//break it! 
	  {
	    auto iBond1(0),iBond2(0),iGlobalNeighbor(0);
	    auto foundValues(false);
	    auto nNeighbors1(dpNeighborsSizes[iOffset + iMonomer]);
	    for (iBond1=0; iBond1< nNeighbors1;iBond1++)
	    {
	      iGlobalNeighbor = dpNeighbors[  iBond1 * rNeighborsPitchElements + iMonomer ];
	      for (iBond2=0; iBond2< dpNeighborsSizes[iGlobalNeighbor]; iBond2++)
	      {
		if (iOffset + iMonomer ==   dpNeighbors[  iBond2 * rNeighborsPitchElements + iGlobalNeighbor-iOffset ])
		{
		  foundValues=true;
		  break;
		}
	      }
	    }
	    if (foundValues)
	    {
	      dpNeighborsSizes[ iGlobalNeighbor ]--;
	      dpNeighbors[  iBond2 * rNeighborsPitchElements + (iGlobalNeighbor-iOffset) ]=dpNeighbors[  dpNeighborsSizes[iGlobalNeighbor ] * rNeighborsPitchElements + (iGlobalNeighbor-iOffset) ];
	      
	      dpNeighborsSizes[iOffset + iMonomer ]--;
	      dpNeighbors[ iBond1 * rNeighborsPitchElements + iMonomer ]=dpNeighbors[ dpNeighborsSizes[iOffset + iMonomer ] * rNeighborsPitchElements + iMonomer ];
	      
	      dBreaks[iMonomer+1] = iGlobalNeighbor+1;
	      
	      texAllowedToMove[iMonomer]=0;
	      texAllowedToMove[iGlobalNeighbor-iOffset]=0;
	    }
	  }
	}
      }
    }
}

template< typename T_UCoordinateCuda > 
void UpdaterGPUScBFM_AA_Breaking<T_UCoordinateCuda>::launch_BreakConnections(
	  const size_t nBlocks, const size_t nThreads, 
	  const size_t iSpecies, const uint64_t seed)
{
  kernel_BreakConnections<<<nBlocks,nThreads,0,mStream>>>
  (
      mNeighborsSortedSizes->gpu, 
      mNeighborsSorted->gpu + mNeighborsSortedInfo.getMatrixOffsetElements( iSpecies ),
      mNeighborsSortedInfo.getMatrixPitchElements( iSpecies ),
      mviSubGroupOffsets[ iSpecies ],
      mnElementsInGroup[ iSpecies ],                       
      seed, 
      mGlobalIterator,
      AAMonomerFlag->gpu,
      dBreaks->gpu
  );
//   CUDA_ERROR(hipDeviceSynchronize());
  mGlobalIterator++;
  tracker.trackBreaks( dBreaks->gpu, nReactiveMonomers+1, miNewToi->gpu, mAge);
 
}
template< typename T_UCoordinateCuda > 
UpdaterGPUScBFM_AA_Breaking<T_UCoordinateCuda>::UpdaterGPUScBFM_AA_Breaking():
BaseClass()  ,
dBreaks(NULL)
{
    /**
     * Log control.
     * Note that "Check" controls not the output, but the actualy checks
     * If a checks needs to always be done, then do that check and declare
     * the output as "Info" log level
     */
    mLog.file( __FILENAME__ );
    mLog.deactivate( "Check"     );
    mLog.deactivate( "Error"     );
    mLog.deactivate( "Info"      );
    mLog.deactivate( "Stats"     );
    mLog.deactivate( "Warning"   );
};
template< typename T_UCoordinateCuda > 
void UpdaterGPUScBFM_AA_Breaking<T_UCoordinateCuda>::destruct(){
      
    DeleteMirroredObject deletePointer;
    deletePointer( dBreaks       , "dBreaks"        );
    if ( deletePointer.nBytesFreed > 0 )
    {
        mLog( "Info" )
            << "Freed a total of "
            << prettyPrintBytes( deletePointer.nBytesFreed )
            << " on GPU and host RAM.\n";
    }
}
template< typename T_UCoordinateCuda > 
UpdaterGPUScBFM_AA_Breaking<T_UCoordinateCuda>::~UpdaterGPUScBFM_AA_Breaking()
{
  this->destruct();    
}

template< typename T_UCoordinateCuda >
void UpdaterGPUScBFM_AA_Breaking<T_UCoordinateCuda>::cleanup()
{
    BaseClass::destruct();
    this->destruct();    
    hipDeviceSynchronize();
    hipProfilerStop();
    
}

template < typename T_UCoordinateCuda >
void UpdaterGPUScBFM_AA_Breaking<T_UCoordinateCuda>::initialize()
{
  BaseClass::initialize();
  
  double mBreakingProbability(exp(-energy));
  std::cout <<"nReactiveMonomers+1 = "<< nReactiveMonomers+1 << "\n";
  dBreaks = new MirroredVector<T_Id>(nReactiveMonomers+1,mStream);
  CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBreakingProbability), &mBreakingProbability, sizeof( mBreakingProbability ) ) );
  mLog("Info") << "Bond energy is " << energy << " which corresponds to a breaking probabilit of " << mBreakingProbability <<"\n" ;

}

template< typename T_UCoordinateCuda  >
void UpdaterGPUScBFM_AA_Breaking< T_UCoordinateCuda >::setBondEnergy(double energy_){energy=energy_;}


template< typename T_UCoordinateCuda  >
void UpdaterGPUScBFM_AA_Breaking< T_UCoordinateCuda >::runSimulationOnGPU
(
    uint32_t const nMonteCarloSteps
)
{
  std::clock_t const t0 = std::clock();
    CUDA_ERROR( hipStreamSynchronize( mStream ) ); // finish e.g. initializations
    CUDA_ERROR( hipMemcpy( mPolymerSystemSortedOld->gpu, mPolymerSystemSorted->gpu, mPolymerSystemSortedOld->nBytes, hipMemcpyDeviceToDevice ) );
    auto const nSpecies = mnElementsInGroup.size();
    AutomaticThreadChooser chooseThreads(nSpecies);
    chooseThreads.initialize(mCudaProps);
    std::vector< uint64_t > nSpeciesChosen( nSpecies ,0 );

    /* run simulation */
    for ( uint32_t iStep = 0; iStep < nMonteCarloSteps; ++iStep, ++mAge )
    {
        if ( mUsePeriodicMonomerSorting && ( mAge % mnStepsBetweenSortings == 0 ) )
        {
            mLog( "Stats" ) << "Resorting at age / step " << mAge << "\n";
//             doSpatialSorting();
        }
        if ( useOverflowChecks )
        {
            /**
             * for uint8_t we have to check for overflows every 127 steps, as
             * for 128 steps we couldn't say whether it actually moved 128 steps
             * or whether it moved 128 steps in the other direction and was wrapped
             * to be equal to the hypothetical monomer above
             */
            auto constexpr boxSizeCudaType = 1ll << ( sizeof( T_UCoordinateCuda ) * CHAR_BIT );
            auto constexpr nStepsBetweenOverflowChecks = boxSizeCudaType / 2 - 1;
            if ( iStep != 0 && iStep % nStepsBetweenOverflowChecks == 0 )
            {
                findAndRemoveOverflows( false );
                CUDA_ERROR( hipMemcpyAsync( mPolymerSystemSortedOld->gpu,
                    mPolymerSystemSorted->gpu, mPolymerSystemSortedOld->nBytes,
                    hipMemcpyDeviceToDevice, mStream ) );
            }
        }
        /* one Monte-Carlo step:
         *  - tries to move on average all particles one time
         *  - each particle could be touched, not just one group */
        for ( uint32_t iSubStep = 0; iSubStep < nSpecies; ++iSubStep ) 
	{
            auto const iStepTotal = iStep * nSpecies + iSubStep;
            auto  iOffsetLatticeTmp = ( iStepTotal % mnLatticeTmpBuffers )
            * ( mBoxX * mBoxY * mBoxZ * sizeof( mLatticeTmp->gpu[0] ));
            if (met.getPacking().getBitPackingOn()) 
                iOffsetLatticeTmp /= CHAR_BIT;
            auto texLatticeTmp = mvtLatticeTmp[ iStepTotal % mnLatticeTmpBuffers ];

            if (met.getPacking().getNBufferedTmpLatticeOn()) {
                    iOffsetLatticeTmp = 0u;
                    texLatticeTmp = mLatticeTmp->texture;
            }
            /* randomly choose which monomer group to advance */
            auto const iSpecies = randomNumbers.r250_rand32() % nSpecies;
            auto const seed     = randomNumbers.r250_rand32();
            auto const nThreads = chooseThreads.getBestThread(iSpecies);
            auto const nBlocks  = ceilDiv( mnElementsInGroup[ iSpecies ], nThreads );
            auto const useCudaMemset = chooseThreads.useCudaMemset(iSpecies);
            chooseThreads.addRecord(iSpecies, mStream);

            nSpeciesChosen[ iSpecies ] += 1;
            
	    if (iSpecies != ChainEndSpecies )
	    {

	      launch_CheckSpecies(nBlocks, nThreads, iSpecies, iOffsetLatticeTmp, seed);

	      if ( useCudaMemset )
		  launch_PerformSpeciesAndApply(nBlocks, nThreads, iSpecies, texLatticeTmp);
	      else
		  launch_PerformSpecies(nBlocks,nThreads,iSpecies,texLatticeTmp);

	      if ( useCudaMemset ){
		  if(met.getPacking().getNBufferedTmpLatticeOn()){
		      /* we only need to delete when buffers will wrap around and
			  * on the last loop, so that on next runSimulationOnGPU
			  * call mLatticeTmp is clean */
		      if ( ( iStepTotal % mnLatticeTmpBuffers == 0 ) ||
			  ( iStep == nMonteCarloSteps-1 && iSubStep == nSpecies-1 ) )
		      {
			  hipMemsetAsync( (void*) mLatticeTmp->gpu, 0, mLatticeTmp->nBytes, mStream );
		      }
		  }else
		      mLatticeTmp->memsetAsync(0);
	      }
	      else
		  launch_ZeroArraySpecies(nBlocks,nThreads,iSpecies);
            }
            else 
            {
	      for(uint32_t n=0; n < 1; n++)
	      {

	      
		launch_CheckReactiveSpecies(nBlocks, nThreads, iSpecies, iOffsetLatticeTmp, seed, n, AAMonomerFlag->texture );
		if ( useCudaMemset )
		    launch_PerformSpeciesAndApply(nBlocks, nThreads, iSpecies, texLatticeTmp);
		else
		    launch_PerformSpecies(nBlocks,nThreads,iSpecies,texLatticeTmp);

		if ( useCudaMemset ){
		    if(met.getPacking().getNBufferedTmpLatticeOn()){
			/* we only need to delete when buffers will wrap around and
			    * on the last loop, so that on next runSimulationOnGPU
			    * call mLatticeTmp is clean */
			if ( ( iStepTotal % mnLatticeTmpBuffers == 0 ) ||
			    ( iStep == nMonteCarloSteps-1 && iSubStep == nSpecies-1 ) )
			{
			    hipMemsetAsync( (void*) mLatticeTmp->gpu, 0, mLatticeTmp->nBytes, mStream );
			}
		    }else
			mLatticeTmp->memsetAsync(0);
		}
		else
		    launch_ZeroArraySpecies(nBlocks,nThreads,iSpecies);
	      }
	      chooseThreads.analyze(iSpecies,mStream);
            }
        } // iSubstep
       
	//here we could again benchmark for a better performance gain...
        auto const nThreads = chooseThreads.getBestThread(ChainEndSpecies);
	auto const nBlocks  = ceilDiv( mnElementsInGroup[ ChainEndSpecies ], nThreads );
	launch_initializeReactiveLattice( nBlocks, nThreads, ChainEndSpecies);
	if (mLog( "Check" ).isActive())
	  checkReactiveLatticeOccupation();
	auto const nThreads_c = 128;
	auto const nBlocks_c  = ceilDiv( nReactiveMonomers, nThreads_c );
	auto const seed     = randomNumbers.r250_rand32();
        launch_CheckConnection(nBlocks_c,nThreads_c, ChainEndSpecies,seed);
	launch_ApplyConnection(nBlocks_c,nThreads_c, ChainEndSpecies);
	launch_resetReactiveLattice( nBlocks, nThreads, ChainEndSpecies);
	//breaks connections
	launch_BreakConnections(nBlocks_c,nThreads_c, ChainEndSpecies, seed);	
    } // iStep
    
    std::clock_t const t1 = std::clock();
    double const dt = float(t1-t0) / CLOCKS_PER_SEC;
    mLog( "Info" )
    << "run time (GPU): " << nMonteCarloSteps << "\n"
    << "mcs = " << nMonteCarloSteps  << "  speed [performed monomer try and move/s] = MCS*N/t: "
    << nMonteCarloSteps * ( mnAllMonomers / dt )  << "     runtime[s]:" << dt << "\n";
    BaseClass::doCopyBack();
    BaseClass::checkSystem(); // no-op if "Check"-level deactivated
    tracker.dumpReactions();    


}


template class UpdaterGPUScBFM_AA_Breaking< uint8_t  >;
template class UpdaterGPUScBFM_AA_Breaking< uint16_t >;
template class UpdaterGPUScBFM_AA_Breaking< uint32_t >;
template class UpdaterGPUScBFM_AA_Breaking<  int16_t >;
template class UpdaterGPUScBFM_AA_Breaking<  int32_t >;


